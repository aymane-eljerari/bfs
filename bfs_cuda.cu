#include "hip/hip_runtime.h"
/*
  To run CUDA program:

     nvcc -arch sm_35 bfs_cuda.cu -o bfs_cuda

     Need to add compiler options: -arch sm_35 (this is cuda compiler dependent)

*/
#include <stdio.h>
#include <stdlib.h>
#include "graph.h"
#include <time.h>


#define BLOCKS 16
#define THREADS 256


__global__ void kernel_cuda_simple(int **v_adj_begin, int **v_adj_list, int **result, int num_vertices, int n)
{
    /* 
        CUDA implementation 
        v_adj_list: concatenation of all adjacency lists of all vertices
        v_adj_begin: array of size V, storing offsets into previous array
        v_adj_length: array of size V, storing length of every adjacency list 
    */
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x

    for (int v = 0; v < num_vertices; v += num_threads)
    {
        int vertex = v + tid;

        if (vertex < num_vertices)
        {
            for (int n = 0; n < v_adj_length[vertex]; n++)
            {
                int neighbor = v_adj_list[v_adj_begin[vertex] + n]

                if (result[neighbor] > result[vertex] + 1)
                {
                    result[neighbor] = result[vertex] + 1;
                    *still_running = true;
                }
                }
        }
    }
}

void run()
{
    bool *still_running = true;

    while (*still_running)
    {
        hipMemcpy(k_still_running, &false_value, sizeof(bool) * 1, hipMemcpyHostToDevice);
        kernel_cuda_simple<<<BLOCKS, THREADS>>>();
        hipMemcpy(still_running, k_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
}


__global__ void bfs_kernel(int *adj_matrix, int *queue, bool *visited, int *distance, int *parent, int source, int level, int num_vertices)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < level)
    {
        int node = queue[tid];
        visited[node] = true;

        for (int neighbor = 0; neighbor < num_vertices; neighbor++)
        {
            if (adj_matrix[node*num_vertices+neighbor] != 0 && !visited[neighbor])
            {
                visited[neighbor] = true;
                distance[neighbor] = distance[node] + 1;
                parent[neighbor] = node;
                queue[level++] = neighbor;
            }
        }
    }
}

__global__ void bfs(int *adj_matrix, int *queue, bool *visited, int *distance, int *parent, int source, int num_vertices)
{
    int level = 0;
    queue[level] = source;
    visited[source] = true;

    while (level < num_vertices)
    {
        int num_blocks = (level + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_kernel<<<num_blocks, BLOCK_SIZE>>>(adj_matrix, queue, visited, distance, parent, source, level, num_vertices);
        hipDeviceSynchronize();
        level += BLOCK_SIZE * num_blocks;
    }
}



int main() {

    // Select GPU
    hipSetDevice(0);

    // GPU Timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

    // Allocate Host Memory
    Graph g;
    int vertexCount = 10000;
    int maxDegree = 5;

    // Initialize Host Memory
    generate(&g, vertexCount, maxDegree);

    size_t allocSize = vertexCount * sizeof(int);

    // Allocate Device Memory
    int *adjMatrix,

    hipMalloc((void **)adjMatrix, allocSize);



    /* GPU Implementation: Global Memory */

    // Create the CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);

    // Initialize Device Memory
    hipMemcpy(adjMatrix, g, allocSize, hipMemcpyHostToDevice);

    dim3 dimGrid(ciel((float)vertexCount/(float)BLOCK_SIZE),ceil((float)vertexCount/(float)BLOCK_SIZE, 1));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    

    /* Kernels */
    // bfs(int *adjacency_list, int *queue, bool *visited, int *distance, int *parent, int source, int num_vertices)
    bfs<<<dimGrid, dimBlock>>>()

    hipMemcpy()
    // Stop and destroy the timer


    // Free the memory
    


    return 0;
}