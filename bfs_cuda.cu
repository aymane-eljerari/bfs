/*
  To run CUDA program:

     nvcc -arch sm_35 bfs_cuda.cu -o bfs_cuda

     Need to add compiler options: -arch sm_35 (this is cuda compiler dependent)

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include "include/graph.h"
#include <time.h>


#define BLOCKS 16
#define THREADS 256



/* Graph Generation Functions */

#define MAX_QUEUE_SIZE 10000000
#define RANDOM_GENERATE 0 // Whether to generate randomly or wrt to SEED.
#define SEED 42

typedef struct {
    int vertexCount;
    int **adjMatrix;
} Graph;

/* Graph operations */
void initGraph(Graph *g, int vertexCount) {
    int i;
    g->vertexCount = vertexCount;
    g->adjMatrix = (int**)malloc(vertexCount * sizeof(int*));
    for (i = 0; i < vertexCount; i++) {
        g->adjMatrix[i] = (int*)calloc(vertexCount, sizeof(int));
    }
}

void addEdge(Graph *g, int src, int dest) {
    g->adjMatrix[src][dest] = 1;
    g->adjMatrix[dest][src] = 1;
}

void generate(Graph *g, int vertexCount, int maxDegree) {
    int i, j;
    initGraph(g, vertexCount);
    
    if (RANDOM_GENERATE){srand(time(NULL));}
    else{srand(SEED);}

    for (i = 0; i < vertexCount; i++) {
        int degree = rand() % maxDegree;
        for (j = 0; j < degree; j++) {
            int dest = rand() % vertexCount;
            if (i != dest && g->adjMatrix[i][dest] == 0) {
                addEdge(g, i, dest);
            }
        }
    }
}

void printAdjacencyMatrix(Graph *g) {
    int i, j, k;

    printf("\nAdjacency Matrix:\n\n   ");  
    for (k = 0; k < g->vertexCount; k++){
        printf("%d ", k);
    }
    printf("   \n  ");
    for (k = 0; k < g->vertexCount; k++){
        printf("--");
    }
    printf("\n");

    for (i = 0; i < g->vertexCount; i++){
        for (j = 0; j < g->vertexCount; j++){
            if(j == 0){
                printf("%d| ", i);
            }
            printf("%d ", g->adjMatrix[i][j]);
        }
        printf("\n");
    }
}

/* Queue Functions */
typedef struct {
    int front, rear;
    int data[MAX_QUEUE_SIZE];
} Queue;

/* Queue operations */
void initQueue(Queue *q) {
    q->front = q->rear = -1;
}

bool isQueueEmpty(Queue *q) {
    return (q->front == -1 && q->rear == -1);
}

bool isQueueFull(Queue *q) {
    return (q->rear == MAX_QUEUE_SIZE - 1);
}

int queueLength(Queue *q) {
    return (q->rear - q->front + 1);
}

void enqueue(Queue *q, int val) {
    if (isQueueFull(q)) {
        printf("Queue overflow\n");
        return;
    } else if (isQueueEmpty(q)) {
        q->front = q->rear = 0;
    } else {
        q->rear++;
    }
    q->data[q->rear] = val;
}

int dequeue(Queue *q) {
    int val;
    if (isQueueEmpty(q)) {
        printf("Queue underflow\n");
        return -1;
    } else if (q->front == q->rear) {
        val = q->data[q->front];
        q->front = q->rear = -1;
    } else {
        val = q->data[q->front];
        q->front++;
    }
    return val;
}

void printQueue(Queue *q) {
    int i;
    printf("Queue: ");
    for (i = q->front; i <= q->rear; i++) {
        printf("%d ", q->data[i]);
    }
    printf("\n");
}

/********************************/

// __global__ void kernel_cuda_simple(int **v_adj_begin, int **v_adj_list, int **result, int num_vertices, int n)
// {
//     /* 
//         CUDA implementation 
//         v_adj_list: concatenation of all adjacency lists of all vertices
//         v_adj_begin: array of size V, storing offsets into previous array
//         v_adj_length: array of size V, storing length of every adjacency list 
//     */
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     int num_threads = blockDim.x * gridDim.x

//     for (int v = 0; v < num_vertices; v += num_threads)
//     {
//         int vertex = v + tid;

//         if (vertex < num_vertices)
//         {
//             for (int n = 0; n < v_adj_length[vertex]; n++)
//             {
//                 int neighbor = v_adj_list[v_adj_begin[vertex] + n]

//                 if (result[neighbor] > result[vertex] + 1)
//                 {
//                     result[neighbor] = result[vertex] + 1;
//                     *still_running = true;
//                 }
//                 }
//         }
//     }
// }

// void run()
// {
//     bool *still_running = true;

//     while (*still_running)
//     {
//         cudaMemcpy(k_still_running, &false_value, sizeof(bool) * 1, cudaMemcpyHostToDevice);
//         kernel_cuda_simple<<<BLOCKS, THREADS>>>();
//         cudaMemcpy(still_running, k_still_running, sizeof(bool) * 1, cudaMemcpyDeviceToHost);
//     }
//     cudaThreadSynchronize();
// }


// __global__ void bfs_kernel(int *adj_matrix, int *queue, bool *visited, int *distance, int *parent, int source, int level, int num_vertices)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     if (tid < level)
//     {
//         int node = queue[tid];
//         visited[node] = true;

//         for (int neighbor = 0; neighbor < num_vertices; neighbor++)
//         {
//             if (adj_matrix[node*num_vertices+neighbor] != 0 && !visited[neighbor])
//             {
//                 visited[neighbor] = true;
//                 distance[neighbor] = distance[node] + 1;
//                 parent[neighbor] = node;
//                 queue[level++] = neighbor;
//             }
//         }
//     }
// }

// __global__ void bfs(int *adj_matrix, int *queue, bool *visited, int *distance, int *parent, int source, int num_vertices)
// {
//     int level = 0;
//     queue[level] = source;
//     visited[source] = true;

//     while (level < num_vertices)
//     {
//         int num_blocks = (level + BLOCK_SIZE - 1) / BLOCK_SIZE;
//         bfs_kernel<<<num_blocks, BLOCK_SIZE>>>(adj_matrix, queue, visited, distance, parent, source, level, num_vertices);
//         cudaDeviceSynchronize();
//         level += BLOCK_SIZE * num_blocks;
//     }
// }


// void bfs_omp(Graph *g, int startVertex, int targetVertex)
// {

//     bool visited[g->vertexCount];
//     int i, j;
//     for (i = 0; i < g->vertexCount; i++) {
//         visited[i] = false;
//     }

//     Queue q;
//     initQueue(&q);

//     visited[startVertex] = true;
//     enqueue(&q, startVertex);

//     bool found = false; // shared variable to indicate whether targetVertex has been found

//     while (!isQueueEmpty(&q)) {
//         int currVertex = dequeue(&q);

//         #pragma omp parallel for shared(found)
//         for (j = 0; j < g->vertexCount; j++) {
//             if (g->adjMatrix[currVertex][j] == 1 && !visited[j]) {
//                 visited[j] = true;
//                 enqueue(&q, j);
//                 if (j == targetVertex) {
//                     found = true;
//                 }
//             }
//         }

//         if (found) {
//             break; // exit the while loop early if targetVertex has been found
//         }
//     }

//     return found;
// }

__global__ void bfs_kernel(Graph *d_graph)
{
    int vertexCount = d_graph->vertexCount;
    int **adjMatrix= d_graph->adjMatrix;

    printf("vertexCount: %d\n", vertexCount);

}



int main() {

    // Select GPU
    hipSetDevice(0);

    // GPU Timing variables
    // cudaEvent_t start, stop;
    // float elapsed_gpu;

    // Allocate Host Memory
    Graph h_graph;
    int vertexCount = 10;
    int maxDegree = 5;

    // Initialize Host Memory
    generate(&h_graph, vertexCount, maxDegree);

    printAdjacencyMatrix(&h_graph);

    // Allocate device memory for the Graph Struct
    Graph *d_graph;
    hipMalloc(&d_graph, sizeof(Graph));

    // Allocate device memory for the array of pointers
    int **d_adjMatrix;
    hipMalloc(&d_adjMatrix, sizeof(int*) * vertexCount);

    // Allocate device memory for the 2D array
    int *d_adjMatrix_data;
    hipMalloc(&d_adjMatrix_data, sizeof(int) * vertexCount * vertexCount);

    // Copy the data to the device memory
    hipMemcpy(d_adjMatrix_data, h_graph.adjMatrix, sizeof(int) * vertexCount * vertexCount, hipMemcpyHostToDevice);
    for (int i = 0; i < vertexCount; i++) {
        hipMemcpy(&d_adjMatrix[i], &d_adjMatrix_data[i * vertexCount], sizeof(int*), hipMemcpyDeviceToHost);
    }

    // Initialize the device graph's adjMatrix pointer with the d_adjMatrix pointer
    hipMemcpy(&d_graph->adjMatrix, &d_adjMatrix, sizeof(int **), hipMemcpyDeviceToHost);

    // Copy host graph to the device graph
    hipMemcpy(d_graph, &h_graph, sizeof(Graph), hipMemcpyHostToDevice);

    // Free device memory for the adjacency matrix
    hipFree(d_adjMatrix_data);



    /* GPU Implementation */

    // Create the CUDA events
    // cudaEventCreate(&start);
    // cudaEventCreate(&stop);

    // cudaEventRecord(start, 0);

    // Initialize Device Memory


    /* Kernel */
    bfs_kernel<<<1,1>>>(d_graph);
    

    // Stop and destroy the timer
    // cudaEventRecord(stop,0);
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&elapsed_gpu, start, stop);
    // printf("\nGPU time: %f (msec)\n", elapsed_gpu);
    // cudaEventDestroy(start);
    // cudaEventDestroy(stop);


    // Free the memory
    hipFree(d_graph);


    return 0;
}