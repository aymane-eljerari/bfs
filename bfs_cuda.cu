/*
  To run CUDA program:

     nvcc -arch sm_35 bfs_cuda.cu -o bfs_cuda

     Need to add compiler options: -arch sm_35 (this is cuda compiler dependent)

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include "include/graph.h"
#include <time.h>


#define BLOCKS 16
#define THREADS 256
#define MAX_VERTEX_COUNT 1000



/* Graph Generation Functions */

#define MAX_QUEUE_SIZE 10000000
#define RANDOM_GENERATE 0 // Whether to generate randomly or wrt to SEED.
#define SEED 42

typedef struct {
    int vertexCount;
    int **adjMatrix;
} Graph;

/* Graph operations */
void initGraph(Graph *g, int vertexCount) {
    int i;
    g->vertexCount = vertexCount;
    g->adjMatrix = (int**)malloc(vertexCount * sizeof(int*));
    for (i = 0; i < vertexCount; i++) {
        g->adjMatrix[i] = (int*)calloc(vertexCount, sizeof(int));
    }
}

void addEdge(Graph *g, int src, int dest) {
    g->adjMatrix[src][dest] = 1;
    g->adjMatrix[dest][src] = 1;
}

void generate(Graph *g, int vertexCount, int maxDegree) {
    int i, j;
    initGraph(g, vertexCount);
    
    if (RANDOM_GENERATE){srand(time(NULL));}
    else{srand(SEED);}

    for (i = 0; i < vertexCount; i++) {
        int degree = rand() % maxDegree;
        for (j = 0; j < degree; j++) {
            int dest = rand() % vertexCount;
            if (i != dest && g->adjMatrix[i][dest] == 0) {
                addEdge(g, i, dest);
            }
        }
    }
}

void printAdjacencyMatrix(Graph *g) {
    int i, j, k;

    printf("\nAdjacency Matrix:\n\n   ");  
    for (k = 0; k < g->vertexCount; k++){
        printf("%d ", k);
    }
    printf("   \n  ");
    for (k = 0; k < g->vertexCount; k++){
        printf("--");
    }
    printf("\n");

    for (i = 0; i < g->vertexCount; i++){
        for (j = 0; j < g->vertexCount; j++){
            if(j == 0){
                printf("%d| ", i);
            }
            printf("%d ", g->adjMatrix[i][j]);
        }
        printf("\n");
    }
}

/* Queue Functions */
typedef struct {
    int front, rear;
    int data[MAX_QUEUE_SIZE];
} Queue;

/* Queue operations */
void initQueue(Queue *q) {
    q->front = q->rear = -1;
}

bool isQueueEmpty(Queue *q) {
    return (q->front == -1 && q->rear == -1);
}

bool isQueueFull(Queue *q) {
    return (q->rear == MAX_QUEUE_SIZE - 1);
}

int queueLength(Queue *q) {
    return (q->rear - q->front + 1);
}

void enqueue(Queue *q, int val) {
    if (isQueueFull(q)) {
        printf("Queue overflow\n");
        return;
    } else if (isQueueEmpty(q)) {
        q->front = q->rear = 0;
    } else {
        q->rear++;
    }
    q->data[q->rear] = val;
}

int dequeue(Queue *q) {
    int val;
    if (isQueueEmpty(q)) {
        printf("Queue underflow\n");
        return -1;
    } else if (q->front == q->rear) {
        val = q->data[q->front];
        q->front = q->rear = -1;
    } else {
        val = q->data[q->front];
        q->front++;
    }
    return val;
}

void printQueue(Queue *q) {
    int i;
    printf("Queue: ");
    for (i = q->front; i <= q->rear; i++) {
        printf("%d ", q->data[i]);
    }
    printf("\n");
}

/********************************/


__device__ bool bfs(Graph *graph, int start, int finish) {
    int vertexCount = graph->vertexCount;
    int **adjMatrix = graph->adjMatrix;

    // Initialize visited and queue arrays
    __shared__ bool visited[MAX_VERTEX_COUNT];
    __shared__ int queue[MAX_VERTEX_COUNT];
    __shared__ int front;
    __shared__ int rear;

    for (int i = threadIdx.x; i < vertexCount; i += blockDim.x) {
        visited[i] = false;
    }

    if (threadIdx.x == 0) {
        visited[start] = true;
        front = -1;
        rear = -1;
        queue[++rear] = start;
    }

    __syncthreads();

    // Perform BFS
    while (true) {
        __syncthreads();

        if (front == rear) {
            // Queue is empty
            break;
        }

        int vertex = -1;

        if (threadIdx.x == 0) {
            // Dequeue a vertex from queue
            vertex = queue[++front];
        }
        printf("vertex: %d\n", vertex);

        __syncthreads();

        if (vertex == finish) {
            // Path found
            return true;
        }
        printf("threadidx: %d\n", threadIdx.x);
        printf("vertexCount: %d\n", vertexCount);
        for (int i = threadIdx.x; i < vertexCount; i += blockDim.x) {
            printf("adjMatrix[vertex][i]: %d\n", adjMatrix[vertex][i]);

            if (adjMatrix[vertex][i] == 1 && !visited[i]) {
                
                visited[i] = true;

                // Enqueue the vertex
                int newRear = atomicAdd(&rear, 1);
                queue[newRear] = i;
            }
        }
    }

    // Path not found
    return false;
}

__global__ void bfs_kernel(Graph *graph, int start, int finish, bool *found) {
    *found = bfs(graph, start, finish);
}



int main() {

    // Select GPU
    hipSetDevice(0);

    // GPU Timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

    // Allocate Host Memory
    Graph h_graph;
    int vertexCount = MAX_VERTEX_COUNT;
    int maxDegree = 10;

    // Initialize Host Memory
    generate(&h_graph, vertexCount, maxDegree);

    // printAdjacencyMatrix(&h_graph);

    // Allocate device memory for the Graph Struct
    Graph *d_graph;
    hipMalloc(&d_graph, sizeof(Graph));

    // Allocate device memory for the array of pointers
    int **d_adjMatrix;
    hipMalloc(&d_adjMatrix, sizeof(int*) * vertexCount);

    // Allocate device memory for the 2D array
    int *d_adjMatrix_data;
    hipMalloc(&d_adjMatrix_data, sizeof(int) * vertexCount * vertexCount);

    // Copy the data to the device memory
    hipMemcpy(d_adjMatrix_data, h_graph.adjMatrix, sizeof(int) * vertexCount * vertexCount, hipMemcpyHostToDevice);
    for (int i = 0; i < vertexCount; i++) {
        hipMemcpy(&d_adjMatrix[i], &d_adjMatrix_data[i * vertexCount], sizeof(int*), hipMemcpyDeviceToHost);
    }

    // Initialize the device graph's adjMatrix pointer with the d_adjMatrix pointer
    hipMemcpy(&d_graph->adjMatrix, &d_adjMatrix, sizeof(int **), hipMemcpyDeviceToHost);

    // Copy host graph to the device graph
    hipMemcpy(d_graph, &h_graph, sizeof(Graph), hipMemcpyHostToDevice);

    // Free device memory for the adjacency matrix
    hipFree(d_adjMatrix_data);


    bool h_found;
    bool *d_found;
    hipMalloc(&d_found, sizeof(bool));


    /* GPU Implementation */

    // Create the CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int source = 5;
    int target = 8;

    /* Kernel */
    bfs_kernel<<<1, 16>>>(d_graph, source, target, d_found);
    // cudaDeviceSynchronize();

    hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);

    if (h_found) {
        printf("Path found from %d to %d\n", source, target);
    } else {
        printf("No path found from %d to %d\n", source, target);
    }
    

    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // Free the memory
    hipFree(d_graph);
    hipFree(d_found);

    return 0;
}